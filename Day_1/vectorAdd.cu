
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vecAdd(const float *a, const float *b, float *c, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n){
        c[i] = a[i] + b[i];
    }

}

int main(){
    const int N = 10;
    float A[N], B[N], C[N];
    for (int i = 0; i < N; ++i) {
        A[i] = static_cast<float>(i);      // A = [0, 1, 2, ..., 9]
        B[i] = static_cast<float>(i * 2);  // B = [0, 2, 4, ..., 18]
    }
    float *d_a, *d_b,*d_c;
    hipMalloc(&d_a,N*sizeof(float));
    hipMalloc(&d_b,N*sizeof(float));
    hipMalloc(&d_c,N*sizeof(float));
    hipMemcpy(d_a,A,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_b,B,N*sizeof(float),hipMemcpyHostToDevice);
    int blocksize=256;
    int gridsize=N + blocksize - 1/blocksize;
    vecAdd<<<gridsize,blocksize>>>(d_a,d_b,d_c,N);
    hipMemcpy(C,d_c,N*sizeof(float),hipMemcpyDeviceToHost);

    std::cout << "Result vector C = A + B:\n";
    for (int i = 0; i < N; ++i) {
        std::cout << A[i] << " + " << B[i] << " = " << C[i] << "\n";
    }
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}