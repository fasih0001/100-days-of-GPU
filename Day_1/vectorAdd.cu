
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vecAdd(const float *a, const float *b, float *c, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n){
        c[i] = a[i] + b[i];
    }

}

int main(){
    const int N = 10;
    float A[N], B[N], C[N];

    float *d_a, *d_b,*d_c;
    hipMalloc(&d_a,N*sizeof(float));
    hipMalloc(&d_b,N*sizeof(float));
    hipMalloc(&d_c,N*sizeof(float));
    hipMemcpy(d_a,A,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_b,B,N*sizeof(float),hipMemcpyHostToDevice);
    int blocksize=256;
    int gridsize=ceil(N/blocksize);
    vecAdd<<<gridsize,blocksize>>>(d_a,d_b,d_c,N);
    hipMemcpy(C,d_c,N*sizeof(float),hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}